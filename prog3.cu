#include "hip/hip_runtime.h"
/*
 * The linear version of the Kuramoto model using a parallel reduction.
 *
 * (C) 2009 Michal Januszewski, Marcin Kostur
 *     Institute of Physics, University of Silesia, Katowice
 *
 * This file is subject to the terms and conditions of the GNU General Public
 * License v3.
 */

#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>

#include <getopt.h>
#include <unistd.h>
#include <time.h>

#include <gsl/gsl_histogram.h>

#include "rng.cu"

#define MAX_STEPS 2000
#define OUTPUT_NTH 100
#define BINS 64

__constant__ float T = 1.0f;
__constant__ float dt = 0.0f;
__constant__ float K = 4.f;
__constant__ float F0 = 0.0f;
__constant__ int num_particles = 0;

__constant__ float S = 0.0f;
__constant__ float C = 0.0f;

int sps = 100;		// steps per second
int niceness = 0;
int particles = 4096;

float hist_xmin = 0.0f;
float hist_xmax = 2.0f * PI;

bool output_histogram = false;
bool output_avg = false;

__global__ void velocities_level0sin(float *in, float *out)
{
	extern __shared__ float sdata[];

	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;
	sdata[tid] = sinf(in[i]) + sinf(in[i+blockDim.x]);
	__syncthreads();

	for (unsigned int s = blockDim.x/2; s > 0; s >>= 1) {
		if (tid < s) {
			sdata[tid] += sdata[tid + s];
		}
		__syncthreads();
	}

	if (tid == 0) out[blockIdx.x] = sdata[0];
}

__global__ void velocities_level0cos(float *in, float *out)
{
	extern __shared__ float sdata[];

	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;
	sdata[tid] = cosf(in[i]) + cosf(in[i+blockDim.x]);
	__syncthreads();

	for (unsigned int s = blockDim.x/2; s > 0; s >>= 1) {
		if (tid < s) {
			sdata[tid] += sdata[tid + s];
		}
		__syncthreads();
	}

	if (tid == 0) out[blockIdx.x] = sdata[0];
}

__global__ void velocities_level1(float *in, float *out)
{
	extern __shared__ float sdata[];

	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;
	sdata[tid] = in[i] + in[i+blockDim.x];
	__syncthreads();

	for (unsigned int s=blockDim.x/2; s > 0; s >>= 1) {
		if (tid < s) {
			sdata[tid] += sdata[tid + s];
		}
		__syncthreads();
	}

	if (tid == 0) out[blockIdx.x] = sdata[0];
}

__global__ void calculateVelocities(float *x, float *v)
{
	int i, j, tile, idx, gidx = blockIdx.x * blockDim.x + threadIdx.x;
	extern __shared__ float shx[];

	float mx = x[gidx];
	float mv = 0.0f;

	for (i = 0, tile = 0; i < num_particles; i += blockDim.x, tile++) {
		idx = tile * blockDim.x + threadIdx.x;
		shx[threadIdx.x] = x[idx];

		__syncthreads();
		for (j = 0; j < blockDim.x; j++) {
			mv += sinf(shx[j] - mx);
		}
		__syncthreads();
	}

	v[gidx] = mv;
}

#define MAX_THREADS 256

void calculateSC(float *x, float *temp, float *cpu_temp)
{
	int i;
	float avgsin = 0.0f, avgcos = 0.0f;
	int blocks = particles/MAX_THREADS/2.0f;

	velocities_level0sin<<<blocks, MAX_THREADS, MAX_THREADS * sizeof(float)>>>(x, temp);
	hipMemcpy(cpu_temp, temp, blocks * sizeof(float), hipMemcpyDeviceToHost);
	for (i = 0; i < blocks; i++) {
		avgsin += cpu_temp[i];
	}

	velocities_level0cos<<<blocks, MAX_THREADS, MAX_THREADS * sizeof(float)>>>(x, temp);
	hipMemcpy(cpu_temp, temp, blocks * sizeof(float), hipMemcpyDeviceToHost);
	for (i = 0; i < blocks; i++) {
		avgcos += cpu_temp[i];
	}

	hipMemcpyToSymbol(HIP_SYMBOL(S), &avgsin, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(C), &avgcos, sizeof(float));
}

__global__ void advanceSystem(unsigned int *rng_state, float *x)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	float lx = x[idx];
	float n1, n2;

	n1 = rng_uni(&rng_state[idx]);
	n2 = rng_uni(&rng_state[idx]);

	bm_trans(n1, n2);

	x[idx] = lx + (F0 + K*(S*cosf(lx) - C*sinf(lx))/num_particles)*dt + sqrtf(2.0 * T * dt) * n1;
}

static struct option options[] = {
	{ "paths", required_argument, NULL, 'p' },
	{ "coupling", required_argument, NULL, 'K' },
	{ "noise", required_argument, NULL, 'T' },
	{ "mode", required_argument, NULL, 0x101 }
};

void usage(char **argv)
{
	printf("Usage: %s <params> [options]\n\n", argv[0]);
	printf("Required parameters:\n");
	printf("  --mode=MODE         MODE can be one of: hist, avg\n");
	printf("                      avg: outputs the sum of all sin(t) and cos(t) terms\n");
	printf("                      hist: outputs a position histogram\n\n");
	printf("Other options:\n");
	printf("  -p, --paths=NUM     set the number of paths to NUM\n");
	printf("  -K, --coupling=NUM  set the coupling constant K\n");
	printf("  -T, --noise=NUM     set the noise strength\n");

	printf("\nEXAMPLE using gnuplot: \n");
	printf("sp '< ./prog3  --paths=1000000 --mode=hist' u 1:2:3 w l  \n");
	printf("  \n");
}

void parse_params(int argc, char **argv)
{
	int c;
	float tmp;
	while ((c = getopt_long(argc, argv, "p:K:T:", options, NULL)) != EOF) {
		switch (c) {
		case 'p':
			particles = (atoi(optarg) / MAX_THREADS) * MAX_THREADS;
			break;
		case 'T':
			tmp = atof(optarg);
			hipMemcpyToSymbol(HIP_SYMBOL(T), &tmp, sizeof(float));
			break;
		case 'K':
			tmp = atof(optarg);
			hipMemcpyToSymbol(HIP_SYMBOL(K), &tmp, sizeof(float));
			break;
		case 0x101:
			if (!strcmp(optarg, "hist"))
				output_histogram = true;
			else if (!strcmp(optarg, "avg"))
				output_avg = true;
			break;
		}
	}
}

int main(int argc, char **argv)
{
	int i, num_threads;

	float *x, *temp;
	float *dx, *dtemp;

	parse_params(argc, argv);

	if (!output_histogram && !output_avg) {
		usage(argv);
		return -1;
	}

	num_threads = particles;
	const int blocksize = 8*64;

	hipMemcpyToSymbol(HIP_SYMBOL(num_particles), &particles, sizeof(int));

	unsigned int *rng_state, *drng_state;
	size_t size = num_threads * sizeof(float);
	size_t size2 = num_threads * sizeof(unsigned int);

	x = (float*)malloc(size);
	temp = (float*)malloc(size);
	rng_state = (unsigned int*)malloc(size2);
	srand(time(0));

	for (i = 0; i < num_threads; i++) {
		x[i] = ((float)random()/RAND_MAX) * 2.0f * PI;
		temp[i] = 0.0f;
	}

	for (i = 0; i < num_threads; i++) {
		rng_state[i] = (unsigned int)random();
	}

	hipMalloc((void**)&drng_state, size2);
	hipMalloc((void**)&dx, size);
	hipMalloc((void**)&dtemp, size);

	hipMemcpy(dx, x, size, hipMemcpyHostToDevice);
	hipMemcpy(dtemp, temp, size, hipMemcpyHostToDevice);
	hipMemcpy(drng_state, rng_state, size2, hipMemcpyHostToDevice);

	float tmp = 1.0 / sps;
	hipMemcpyToSymbol(HIP_SYMBOL(dt), &tmp, sizeof(float));

	gsl_histogram *h = gsl_histogram_alloc(BINS);
	gsl_histogram_set_ranges_uniform(h,hist_xmin,hist_xmax);

	for (long step = 0; step < MAX_STEPS; step++) {
		float t = step / (float)sps;

		calculateSC(dx, dtemp, temp);
		advanceSystem<<<num_threads/blocksize, blocksize>>>(drng_state, dx);

		if ((step == 0 || (step+1) % OUTPUT_NTH == 0) && (output_histogram || output_avg)) {
			hipMemcpy(x, dx, size, hipMemcpyDeviceToHost);
			gsl_histogram_reset(h);

			double s = 0,c = 0;
			for (i = 0; i < num_threads; i++) {
				float tt = fmodf(x[i], (2.0f * PI));
				if (tt < 0.0f) {
					tt += 2.0f * PI;
				}
				s += sin(tt);
				c += cos(tt);
				gsl_histogram_increment(h, tt);
			}
			s /= num_threads;
			c /= num_threads;

			long hist_norm=0;
			if (output_histogram) {
				for (i = 0; i < BINS; i++) {
					hist_norm += gsl_histogram_get(h, i);
				}
				for (i = 0; i < BINS; i++) {
					printf("%f %d %f\n", t, i,
						BINS*gsl_histogram_get(h, i)/(hist_norm*(hist_xmax-hist_xmin)));
				}
				printf("\n");
			}

			if (output_avg) {
				printf("%f %f %f %f\n", t, s, c, sqrt(s*s + c*c));
			}
		}
	}

	free(x);
	free(temp);
	free(rng_state);

	hipFree(drng_state);
	hipFree(dx);
	hipFree(dtemp);
}
